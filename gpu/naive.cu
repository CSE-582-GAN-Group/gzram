#include "hip/hip_runtime.h"
#include "naive.cuh"

#include <hip/hip_runtime.h>
#include "nvcomp/lz4.h"

// Error checking helper for CUDA calls
#define CHECK_CUDA(call)                                                     \
    do                                                                       \
    {                                                                        \
        hipError_t err = call;                                              \
        if (err != hipSuccess)                                              \
        {                                                                    \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err));                                \
            return CUDA_ERROR;                                               \
        }                                                                    \
    } while (0)

// Helper function to initialize empty CompressedData
CompressedData *create_compressed_data(size_t num_pages)
{
    CompressedData *data = (CompressedData *)malloc(sizeof(CompressedData));
    if (!data)
        return NULL;

    data->compressed_pages = (CompressedPage *)malloc(num_pages * sizeof(CompressedPage));
    if (!data->compressed_pages)
    {
        free(data);
        return NULL;
    }

    data->num_pages = num_pages;
    data->original_size = 0;

    // Initialize all chunks to NULL
    for (size_t i = 0; i < num_pages; i++)
    {
        data->compressed_pages[i].data = NULL;
        data->compressed_pages[i].size = 0;
    }

    return data;
}

// New helper function to create CompressedData from existing arrays
CompressedData *create_compressed_data_from_arrays(
    const char **page_data_array,   // Array of pointers to compressed page data
    const size_t *page_sizes_array, // Array of compressed page sizes
    size_t num_pages,               // Number of pages
    size_t original_size            // Original uncompressed data size
)
{
    CompressedData *data = (CompressedData *)malloc(sizeof(CompressedData));
    if (!data)
        return NULL;

    data->compressed_pages = (CompressedPage *)malloc(num_pages * sizeof(CompressedPage));
    if (!data->compressed_pages)
    {
        free(data);
        return NULL;
    }

    data->num_pages = num_pages;
    data->original_size = original_size;

    // Copy each page's data
    for (size_t i = 0; i < num_pages; i++)
    {
        data->compressed_pages[i].size = page_sizes_array[i];
        data->compressed_pages[i].data = (char *)malloc(page_sizes_array[i]);

        if (!data->compressed_pages[i].data)
        {
            // Cleanup on failure
            for (size_t j = 0; j < i; j++)
            {
                free(data->compressed_pages[j].data);
            }
            free(data->compressed_pages);
            free(data);
            return NULL;
        }
        memcpy(data->compressed_pages[i].data, page_data_array[i], page_sizes_array[i]);
    }

    return data;
}

// Create CompressedData that references existing arrays without copying
CompressedData *create_compressed_data_with_references(
    const char **page_data_array,   // Array of pointers to compressed page data
    const size_t *page_sizes_array, // Array of compressed page sizes
    size_t num_pages,               // Number of pages
    size_t original_size            // Original uncompressed data size
)
{
    CompressedData *data = (CompressedData *)malloc(sizeof(CompressedData));
    if (!data)
        return NULL;

    data->compressed_pages = (CompressedPage *)malloc(num_pages * sizeof(CompressedPage));
    if (!data->compressed_pages)
    {
        free(data);
        return NULL;
    }

    data->num_pages = num_pages;
    data->original_size = original_size;

    // Store references to each page's data instead of copying
    for (size_t i = 0; i < num_pages; i++)
    {
        data->compressed_pages[i].size = page_sizes_array[i];
        // Simply store the pointer to the original data
        data->compressed_pages[i].data = (char *)page_data_array[i];
    }

    return data;
}

// Helper function to free CompressedData
void free_compressed_data(CompressedData *data)
{
    if (!data)
        return;

    if (data->compressed_pages)
    {
        for (size_t i = 0; i < data->num_pages; i++)
        {
            free(data->compressed_pages[i].data);
        }
        free(data->compressed_pages);
    }

    free(data);
}

// Initialize CUDA
void cuda_initialize()
{
    hipFree(0);
}

void cuda_free(void *ptr) {
  hipFree(ptr);
}

ErrorCode compress_pipelined(const char *input_data, size_t in_bytes,
                             CompressedData **output)
{
#ifdef DEBUG
    printf("\n=== Pipelined Compression ===\n");
#endif
    int num_batches = 5;
    int num_streams = 3;

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // printf("\n=== Pipelined Performance Profile ===\n");

    // Phase 1: Initial Setup
    hipEventRecord(start);
    const size_t num_pages = (in_bytes + PAGE_SIZE - 1) / PAGE_SIZE;
    const size_t pages_per_batch = (num_pages + num_batches - 1) / num_batches;

    CompressedData *compressed_result = create_compressed_data(num_pages);
    if (!compressed_result)
    {
        return MEMORY_ERROR;
    }
    compressed_result->original_size = in_bytes;

    // Create stream array
    hipStream_t *streams = (hipStream_t *)malloc(num_streams * sizeof(hipStream_t));
    for (int i = 0; i < num_streams; i++)
    {
        CHECK_CUDA(hipStreamCreate(&streams[i]));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("1. Initial Setup: %.3f ms\n", milliseconds);

    // Phase 2: Resource Allocation
    hipEventRecord(start);

    // Allocate device resources for each stream
    struct StreamResources
    {
        char *device_input_data;
        size_t *device_uncompressed_numbytes;
        void **device_uncompressed_data;
        void *device_temp_ptr;
        void **device_compressed_data;
        size_t *device_compressed_numbytes;
        void *compressed_data_buffer;
    };

    StreamResources *resources = (StreamResources *)malloc(num_streams * sizeof(StreamResources));

    size_t max_batch_bytes = pages_per_batch * PAGE_SIZE;
    size_t temp_bytes;
    nvcompBatchedLZ4CompressGetTempSize(pages_per_batch, PAGE_SIZE,
                                        nvcompBatchedLZ4DefaultOpts, &temp_bytes);

    size_t max_out_bytes;
    nvcompBatchedLZ4CompressGetMaxOutputChunkSize(PAGE_SIZE,
                                                  nvcompBatchedLZ4DefaultOpts,
                                                  &max_out_bytes);

    // Allocate resources for each stream
    for (int i = 0; i < num_streams; i++)
    {
        CHECK_CUDA(hipMalloc(&resources[i].device_input_data, max_batch_bytes));
        CHECK_CUDA(hipMalloc(&resources[i].device_uncompressed_numbytes,
                              sizeof(size_t) * pages_per_batch));
        CHECK_CUDA(hipMalloc(&resources[i].device_uncompressed_data,
                              sizeof(void *) * pages_per_batch));
        CHECK_CUDA(hipMalloc(&resources[i].device_temp_ptr, temp_bytes));
        CHECK_CUDA(hipMalloc(&resources[i].device_compressed_data,
                              sizeof(void *) * pages_per_batch));
        CHECK_CUDA(hipMalloc(&resources[i].device_compressed_numbytes,
                              sizeof(size_t) * pages_per_batch));
        CHECK_CUDA(hipMalloc(&resources[i].compressed_data_buffer,
                              max_out_bytes * pages_per_batch));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("2. Resource Allocation: %.3f ms\n", milliseconds);

    // Phase 3: Pipelined Compression
    hipEventRecord(start);

    for (size_t batch = 0; batch < num_batches; batch++)
    {
        int stream_idx = batch % num_streams;
        size_t start_page = batch * pages_per_batch;
        size_t batch_pages = min(pages_per_batch, num_pages - start_page);
        size_t batch_bytes = min(max_batch_bytes, in_bytes - (start_page * PAGE_SIZE));

        // Setup batch metadata
        void **host_uncompressed_data = (void **)malloc(batch_pages * sizeof(void *));
        size_t *host_uncompressed_sizes = (size_t *)malloc(batch_pages * sizeof(size_t));
        void **host_compressed_data = (void **)malloc(batch_pages * sizeof(void *));

        for (size_t i = 0; i < batch_pages; i++)
        {
            size_t page_offset = (start_page + i) * PAGE_SIZE;
            host_uncompressed_sizes[i] = min((unsigned long)PAGE_SIZE, in_bytes - page_offset);
            host_uncompressed_data[i] = resources[stream_idx].device_input_data +
                                        (i * PAGE_SIZE);
            host_compressed_data[i] = (char *)resources[stream_idx].compressed_data_buffer +
                                      (i * max_out_bytes);
        }

        // Copy input data and metadata
        CHECK_CUDA(hipMemcpyAsync(resources[stream_idx].device_input_data,
                                   input_data + (start_page * PAGE_SIZE),
                                   batch_bytes, hipMemcpyHostToDevice,
                                   streams[stream_idx]));

        CHECK_CUDA(hipMemcpyAsync(resources[stream_idx].device_uncompressed_numbytes,
                                   host_uncompressed_sizes,
                                   sizeof(size_t) * batch_pages,
                                   hipMemcpyHostToDevice, streams[stream_idx]));

        CHECK_CUDA(hipMemcpyAsync(resources[stream_idx].device_uncompressed_data,
                                   host_uncompressed_data,
                                   sizeof(void *) * batch_pages,
                                   hipMemcpyHostToDevice, streams[stream_idx]));

        CHECK_CUDA(hipMemcpyAsync(resources[stream_idx].device_compressed_data,
                                   host_compressed_data,
                                   sizeof(void *) * batch_pages,
                                   hipMemcpyHostToDevice, streams[stream_idx]));

        // Compress batch
        nvcompStatus_t comp_res = nvcompBatchedLZ4CompressAsync(
            resources[stream_idx].device_uncompressed_data,
            resources[stream_idx].device_uncompressed_numbytes,
            PAGE_SIZE, batch_pages,
            resources[stream_idx].device_temp_ptr,
            temp_bytes,
            resources[stream_idx].device_compressed_data,
            resources[stream_idx].device_compressed_numbytes,
            nvcompBatchedLZ4DefaultOpts,
            streams[stream_idx]);

        if (comp_res != nvcompSuccess)
        {
            // Cleanup and return error
            for (int i = 0; i < num_streams; i++)
            {
                hipStreamDestroy(streams[i]);
                hipFree(resources[i].device_input_data);
                hipFree(resources[i].device_uncompressed_numbytes);
                hipFree(resources[i].device_uncompressed_data);
                hipFree(resources[i].device_temp_ptr);
                hipFree(resources[i].device_compressed_data);
                hipFree(resources[i].device_compressed_numbytes);
                hipFree(resources[i].compressed_data_buffer);
            }
            free(streams);
            free(resources);
            free_compressed_data(compressed_result);
            return NVCOMP_ERROR;
        }

        // Allocate host buffer for compressed data
        char *host_compressed_buffer;
        CHECK_CUDA(hipHostMalloc(&host_compressed_buffer,
                                  max_out_bytes * batch_pages));

        // Get compressed sizes
        size_t *compressed_sizes = (size_t *)malloc(batch_pages * sizeof(size_t));
        CHECK_CUDA(hipMemcpyAsync(compressed_sizes,
                                   resources[stream_idx].device_compressed_numbytes,
                                   sizeof(size_t) * batch_pages,
                                   hipMemcpyDeviceToHost, streams[stream_idx]));

        // Copy compressed data
        CHECK_CUDA(hipMemcpyAsync(host_compressed_buffer,
                                   resources[stream_idx].compressed_data_buffer,
                                   max_out_bytes * batch_pages,
                                   hipMemcpyDeviceToHost, streams[stream_idx]));

        // Wait for stream operations to complete
        CHECK_CUDA(hipStreamSynchronize(streams[stream_idx]));

        // Store results
        for (size_t i = 0; i < batch_pages; i++)
        {
            size_t page_idx = start_page + i;
            compressed_result->compressed_pages[page_idx].size = compressed_sizes[i];
            compressed_result->compressed_pages[page_idx].data =
                (char *)malloc(compressed_sizes[i]);

            if (!compressed_result->compressed_pages[page_idx].data)
            {
                hipHostFree(host_compressed_buffer);
                free(compressed_sizes);
                free(host_uncompressed_data);
                free(host_uncompressed_sizes);
                free(host_compressed_data);
                return MEMORY_ERROR;
            }

            memcpy(compressed_result->compressed_pages[page_idx].data,
                   host_compressed_buffer + (i * max_out_bytes),
                   compressed_sizes[i]);
        }

        // Cleanup batch resources
        hipHostFree(host_compressed_buffer);
        free(compressed_sizes);
        free(host_uncompressed_data);
        free(host_uncompressed_sizes);
        free(host_compressed_data);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("3. Pipelined Compression: %.3f ms\n", milliseconds);

    // Phase 4: Cleanup
    hipEventRecord(start);

    for (int i = 0; i < num_streams; i++)
    {
        hipStreamDestroy(streams[i]);
        hipFree(resources[i].device_input_data);
        hipFree(resources[i].device_uncompressed_numbytes);
        hipFree(resources[i].device_uncompressed_data);
        hipFree(resources[i].device_temp_ptr);
        hipFree(resources[i].device_compressed_data);
        hipFree(resources[i].device_compressed_numbytes);
        hipFree(resources[i].compressed_data_buffer);
    }

    free(streams);
    free(resources);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("4. Cleanup: %.3f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    *output = compressed_result;
    return SUCCESS;
}

ErrorCode compress_improved_naive(const char *input_data, size_t in_bytes, CompressedData **output)
{
//    printf("\n=== Improved Naive Compression ===\n");

    // Create timing events
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // printf("\n=== Performance Profile ===\n");

    // Phase 1: Initialization
    hipEventRecord(start);

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    const size_t num_pages = (in_bytes + PAGE_SIZE - 1) / PAGE_SIZE;
    // Create output structure
    CompressedData *compressed_result = create_compressed_data(num_pages);
    if (!compressed_result)
    {
        hipStreamDestroy(stream);
        return MEMORY_ERROR;
    }
    compressed_result->original_size = in_bytes;

    // Allocate device input data
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("1. Initialization: %.3f ms\n", milliseconds);

    // Phase 2: Device Memory Allocation & Initial Transfer
    hipEventRecord(start);

    char *device_input_data;
    if (hipMalloc(&device_input_data, in_bytes) != hipSuccess)
    {
        free_compressed_data(compressed_result);
        hipStreamDestroy(stream);
        return CUDA_ERROR;
    }

    // Copy input data to device
    CHECK_CUDA(hipMemcpyAsync(device_input_data, input_data, in_bytes,
                               hipMemcpyHostToDevice, stream));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("2. Initial Device Allocation & Transfer: %.3f ms\n", milliseconds);

    // Phase 3: Page Management Setup
    hipEventRecord(start);

    // Find bytes per page on input data
    size_t *host_uncompressed_numbytes_per_page;
    CHECK_CUDA(hipHostMalloc(&host_uncompressed_numbytes_per_page, sizeof(size_t) * num_pages));
    for (size_t i = 0; i < num_pages; ++i)
    {
        if (i + 1 < num_pages)
        {
            host_uncompressed_numbytes_per_page[i] = PAGE_SIZE;
        }
        else
        {
            host_uncompressed_numbytes_per_page[i] = in_bytes - (PAGE_SIZE * i);
        }
    }

    // Fill in the pointers to the input data for each page
    void **host_uncompressed_data_per_page;
    CHECK_CUDA(hipHostMalloc(&host_uncompressed_data_per_page, sizeof(void *) * num_pages));
    for (size_t i = 0; i < num_pages; ++i)
    {
        host_uncompressed_data_per_page[i] = device_input_data + PAGE_SIZE * i;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("3. Page Management Setup: %.3f ms\n", milliseconds);

    // Phase 4: Device Memory Setup for Pages
    hipEventRecord(start);

    // Copy pointers of sizes and data to device
    size_t *device_uncompressed_numbytes_per_page;
    void **device_uncompressed_data_per_page;
    CHECK_CUDA(hipMalloc(&device_uncompressed_numbytes_per_page, sizeof(size_t) * num_pages));
    CHECK_CUDA(hipMalloc(&device_uncompressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMemcpyAsync(device_uncompressed_numbytes_per_page, host_uncompressed_numbytes_per_page,
                               sizeof(size_t) * num_pages, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(device_uncompressed_data_per_page, host_uncompressed_data_per_page,
                               sizeof(void *) * num_pages, hipMemcpyHostToDevice, stream));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("4. Device Page Setup: %.3f ms\n", milliseconds);

    // Phase 5: Compression Buffer Setup - OPTIMIZED
    hipEventRecord(start);

    size_t temp_bytes;
    nvcompBatchedLZ4CompressGetTempSize(num_pages, PAGE_SIZE,
                                        nvcompBatchedLZ4DefaultOpts, &temp_bytes);
    void *device_temp_ptr;
    CHECK_CUDA(hipMalloc(&device_temp_ptr, temp_bytes));

    // Allocate a single large buffer instead of many small ones
    void **host_compressed_data_per_page;
    void **device_compressed_data_per_page;
    size_t *device_compressed_numbytes_per_page;
    CHECK_CUDA(hipHostMalloc(&host_compressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMalloc(&device_compressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMalloc(&device_compressed_numbytes_per_page, sizeof(size_t) * num_pages));

    size_t max_out_bytes;
    nvcompBatchedLZ4CompressGetMaxOutputChunkSize(PAGE_SIZE,
                                                  nvcompBatchedLZ4DefaultOpts,
                                                  &max_out_bytes);

    // Allocate one large buffer for all pages
    void *compressed_data_buffer;
    CHECK_CUDA(hipMalloc(&compressed_data_buffer, max_out_bytes * num_pages));

    // Set up pointers into the buffer
    for (size_t i = 0; i < num_pages; ++i)
    {
        host_compressed_data_per_page[i] = (char *)compressed_data_buffer + (i * max_out_bytes);
    }
    CHECK_CUDA(hipMemcpyAsync(device_compressed_data_per_page, host_compressed_data_per_page,
                               sizeof(void *) * num_pages, hipMemcpyHostToDevice, stream));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("5. Compression Buffer Setup: %.3f ms\n", milliseconds);

    // Phase 6: Main Compression
    hipEventRecord(start);

    // Compress the data
    nvcompStatus_t comp_res = nvcompBatchedLZ4CompressAsync(
        device_uncompressed_data_per_page,
        device_uncompressed_numbytes_per_page,
        PAGE_SIZE,
        num_pages,
        device_temp_ptr,
        temp_bytes,
        device_compressed_data_per_page,
        device_compressed_numbytes_per_page,
        nvcompBatchedLZ4DefaultOpts,
        stream);

    if (comp_res != nvcompSuccess)
    {
        hipFree(device_input_data);
        hipHostFree(host_uncompressed_numbytes_per_page);
        hipHostFree(host_uncompressed_data_per_page);
        hipFree(device_uncompressed_numbytes_per_page);
        hipFree(device_uncompressed_data_per_page);
        hipFree(device_temp_ptr);
        for (size_t i = 0; i < num_pages; i++)
        {
            hipFree(host_compressed_data_per_page[i]);
        }
        hipHostFree(host_compressed_data_per_page);
        hipFree(device_compressed_data_per_page);
        hipFree(device_compressed_numbytes_per_page);
        free_compressed_data(compressed_result);
        hipStreamDestroy(stream);
        return NVCOMP_ERROR;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("6. Main Compression: %.3f ms\n", milliseconds);

    // Phase 7: Result Collection - OPTIMIZED
    hipEventRecord(start);

    // Allocate a single host buffer for all compressed data
    char *host_compressed_buffer;
    CHECK_CUDA(hipHostMalloc(&host_compressed_buffer, max_out_bytes * num_pages));

    // Get all compressed sizes in one transfer
    size_t *compressed_sizes = (size_t *)malloc(num_pages * sizeof(size_t));
    if (!compressed_sizes)
    {
        return MEMORY_ERROR;
    }

    CHECK_CUDA(hipMemcpyAsync(compressed_sizes, device_compressed_numbytes_per_page,
                               sizeof(size_t) * num_pages, hipMemcpyDeviceToHost, stream));

    // Copy all compressed data in one large transfer
    CHECK_CUDA(hipMemcpyAsync(host_compressed_buffer, compressed_data_buffer,
                               max_out_bytes * num_pages, hipMemcpyDeviceToHost, stream));

    // Wait for transfers to complete
    CHECK_CUDA(hipStreamSynchronize(stream));

    // Set up the output structure (now using CPU memory)
    for (size_t i = 0; i < num_pages; i++)
    {
        compressed_result->compressed_pages[i].size = compressed_sizes[i];
        compressed_result->compressed_pages[i].data = (char *)malloc(compressed_sizes[i]);
        if (!compressed_result->compressed_pages[i].data)
        {
            free(compressed_sizes);
            hipHostFree(host_compressed_buffer);
            return MEMORY_ERROR;
        }

        // Copy from pinned buffer to final destination (CPU memory copy, no CUDA involved)
        memcpy(compressed_result->compressed_pages[i].data,
               host_compressed_buffer + (i * max_out_bytes),
               compressed_sizes[i]);
    }

    free(compressed_sizes);
    hipHostFree(host_compressed_buffer);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("7. Result Collection: %.3f ms\n", milliseconds);

    // Phase 8: Cleanup - OPTIMIZED
    hipEventRecord(start);

    // Single large free instead of many small ones
    hipFree(compressed_data_buffer);
    hipFree(device_input_data);
    hipHostFree(host_uncompressed_numbytes_per_page);
    hipHostFree(host_uncompressed_data_per_page);
    hipFree(device_uncompressed_numbytes_per_page);
    hipFree(device_uncompressed_data_per_page);
    hipFree(device_temp_ptr);
    hipHostFree(host_compressed_data_per_page);
    hipFree(device_compressed_data_per_page);
    hipFree(device_compressed_numbytes_per_page);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("8. Cleanup: %.3f ms\n", milliseconds);

    // Cleanup timing events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipStreamDestroy(stream);

    *output = compressed_result;
    return SUCCESS;
}

ErrorCode compress_naive(const char *input_data, size_t in_bytes, CompressedData **output)
{
//    printf("\n=== Naive Compression ===\n");
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    const size_t num_pages = (in_bytes + PAGE_SIZE - 1) / PAGE_SIZE;

    // Create output structure
    CompressedData *compressed_result = create_compressed_data(num_pages);
    if (!compressed_result)
    {
        hipStreamDestroy(stream);
        return MEMORY_ERROR;
    }

    compressed_result->original_size = in_bytes;

    // Allocate device input data
    char *device_input_data;
    if (hipMalloc(&device_input_data, in_bytes) != hipSuccess)
    {
        free_compressed_data(compressed_result);
        hipStreamDestroy(stream);
        return CUDA_ERROR;
    }

    // Copy input data to device
    CHECK_CUDA(hipMemcpyAsync(device_input_data, input_data, in_bytes,
                               hipMemcpyHostToDevice, stream));

    // Find bytes per page on input data
    size_t *host_uncompressed_numbytes_per_page;
    CHECK_CUDA(hipHostMalloc(&host_uncompressed_numbytes_per_page, sizeof(size_t) * num_pages));
    for (size_t i = 0; i < num_pages; ++i)
    {
        if (i + 1 < num_pages)
        {
            host_uncompressed_numbytes_per_page[i] = PAGE_SIZE;
        }
        else
        {
            host_uncompressed_numbytes_per_page[i] = in_bytes - (PAGE_SIZE * i);
        }
    }

    // Fill in the pointers to the input data for each page
    void **host_uncompressed_data_per_page;
    CHECK_CUDA(hipHostMalloc(&host_uncompressed_data_per_page, sizeof(void *) * num_pages));
    for (size_t i = 0; i < num_pages; ++i)
    {
        host_uncompressed_data_per_page[i] = device_input_data + PAGE_SIZE * i;
    }

    // Copy pointers of sizes and data to device
    size_t *device_uncompressed_numbytes_per_page;
    void **device_uncompressed_data_per_page;
    CHECK_CUDA(hipMalloc(&device_uncompressed_numbytes_per_page, sizeof(size_t) * num_pages));
    CHECK_CUDA(hipMalloc(&device_uncompressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMemcpyAsync(device_uncompressed_numbytes_per_page, host_uncompressed_numbytes_per_page,
                               sizeof(size_t) * num_pages, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(device_uncompressed_data_per_page, host_uncompressed_data_per_page,
                               sizeof(void *) * num_pages, hipMemcpyHostToDevice, stream));

    // Calculate temp data buffer size required for compression and allocate it
    size_t temp_bytes;
    nvcompBatchedLZ4CompressGetTempSize(num_pages, PAGE_SIZE,
                                        nvcompBatchedLZ4DefaultOpts, &temp_bytes);
    void *device_temp_ptr;
    CHECK_CUDA(hipMalloc(&device_temp_ptr, temp_bytes));

    // Allocate space for compressed data
    void **host_compressed_data_per_page;
    void **device_compressed_data_per_page;
    size_t *device_compressed_numbytes_per_page;
    CHECK_CUDA(hipHostMalloc(&host_compressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMalloc(&device_compressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMalloc(&device_compressed_numbytes_per_page, sizeof(size_t) * num_pages));

    size_t max_out_bytes;
    nvcompBatchedLZ4CompressGetMaxOutputChunkSize(PAGE_SIZE,
                                                  nvcompBatchedLZ4DefaultOpts,
                                                  &max_out_bytes);
    for (size_t i = 0; i < num_pages; ++i)
    {
        CHECK_CUDA(hipMalloc(&host_compressed_data_per_page[i], max_out_bytes));
    }
    CHECK_CUDA(hipMemcpyAsync(device_compressed_data_per_page, host_compressed_data_per_page,
                               sizeof(void *) * num_pages, hipMemcpyHostToDevice, stream));

    // Compress the data
    nvcompStatus_t comp_res = nvcompBatchedLZ4CompressAsync(
        device_uncompressed_data_per_page,
        device_uncompressed_numbytes_per_page,
        PAGE_SIZE,
        num_pages,
        device_temp_ptr,
        temp_bytes,
        device_compressed_data_per_page,
        device_compressed_numbytes_per_page,
        nvcompBatchedLZ4DefaultOpts,
        stream);

    if (comp_res != nvcompSuccess)
    {
        hipFree(device_input_data);
        hipHostFree(host_uncompressed_numbytes_per_page);
        hipHostFree(host_uncompressed_data_per_page);
        hipFree(device_uncompressed_numbytes_per_page);
        hipFree(device_uncompressed_data_per_page);
        hipFree(device_temp_ptr);
        for (size_t i = 0; i < num_pages; i++)
        {
            hipFree(host_compressed_data_per_page[i]);
        }
        hipHostFree(host_compressed_data_per_page);
        hipFree(device_compressed_data_per_page);
        hipFree(device_compressed_numbytes_per_page);
        free_compressed_data(compressed_result);
        hipStreamDestroy(stream);
        return NVCOMP_ERROR;
    }

    // Get the compressed sizes and copy compressed data
    size_t *compressed_sizes = (size_t *)malloc(num_pages * sizeof(size_t));
    if (!compressed_sizes)
    {
        // TODO: cleanup resources
        return MEMORY_ERROR;
    }

    CHECK_CUDA(hipMemcpy(compressed_sizes, device_compressed_numbytes_per_page,
                          sizeof(size_t) * num_pages, hipMemcpyDeviceToHost));

    // Copy each compressed chunk
    for (size_t i = 0; i < num_pages; i++)
    {
        compressed_result->compressed_pages[i].size = compressed_sizes[i];
        compressed_result->compressed_pages[i].data = (char *)malloc(compressed_sizes[i]);
        if (!compressed_result->compressed_pages[i].data)
        {
            free(compressed_sizes);
            free_compressed_data(compressed_result);
            // TODO: cleanup CUDA resources
            return MEMORY_ERROR;
        }

        CHECK_CUDA(hipMemcpy(compressed_result->compressed_pages[i].data,
                              host_compressed_data_per_page[i],
                              compressed_sizes[i],
                              hipMemcpyDeviceToHost));
    }

    free(compressed_sizes);

    // Cleanup CUDA resources
    hipFree(device_input_data);
    hipHostFree(host_uncompressed_numbytes_per_page);
    hipHostFree(host_uncompressed_data_per_page);
    hipFree(device_uncompressed_numbytes_per_page);
    hipFree(device_uncompressed_data_per_page);
    hipFree(device_temp_ptr);
    for (size_t i = 0; i < num_pages; i++)
    {
        hipFree(host_compressed_data_per_page[i]);
    }
    hipHostFree(host_compressed_data_per_page);
    hipFree(device_compressed_data_per_page);
    hipFree(device_compressed_numbytes_per_page);
    hipStreamDestroy(stream);

    *output = compressed_result;
    return SUCCESS;
}

ErrorCode decompress_pipelined(const CompressedData *compressed_data, char **output_data, size_t *output_size)
{
    printf("\n=== Pipelined Decompression ===\n");
    const int NUM_STREAMS = 3;  // Number of concurrent streams
    const int NUM_BATCHES = 10; // Number of batches to split the original data into

    hipStream_t streams[NUM_STREAMS];
    hipEvent_t events[NUM_STREAMS];

    // Create streams and events
    for (int i = 0; i < NUM_STREAMS; i++)
    {
        CHECK_CUDA(hipStreamCreate(&streams[i]));
        CHECK_CUDA(hipEventCreate(&events[i]));
    }

    size_t num_pages = compressed_data->num_pages;
    *output_size = compressed_data->original_size;

    // Calculate pages per batch
    size_t pages_per_batch = (num_pages + NUM_BATCHES - 1) / NUM_BATCHES;

    // Allocate unified memory for output
    CHECK_CUDA(hipMallocManaged(output_data, compressed_data->original_size));

    // Process each batch
    for (size_t batch = 0; batch < NUM_BATCHES; batch++)
    {
        size_t batch_start = batch * pages_per_batch;
        size_t batch_end = min(batch_start + pages_per_batch, num_pages);
        size_t batch_pages = batch_end - batch_start;

        if (batch_pages == 0)
            break;

        // Calculate total compressed size for this batch
        size_t batch_compressed_size = 0;
        for (size_t i = batch_start; i < batch_end; i++)
        {
            batch_compressed_size += compressed_data->compressed_pages[i].size;
        }

        // Pages per stream for this batch
        size_t pages_per_stream = (batch_pages + NUM_STREAMS - 1) / NUM_STREAMS;

        // Process the batch using multiple streams
        for (int stream_idx = 0; stream_idx < NUM_STREAMS; stream_idx++)
        {
            size_t stream_start = batch_start + (stream_idx * pages_per_stream);
            size_t stream_end = min(stream_start + pages_per_stream, batch_end);
            size_t stream_pages = stream_end - stream_start;

            if (stream_pages == 0)
                continue;

            // Calculate compressed size for this stream's portion
            size_t stream_compressed_size = 0;
            for (size_t i = stream_start; i < stream_end; i++)
            {
                stream_compressed_size += compressed_data->compressed_pages[i].size;
            }

            // Allocate stream resources
            char *stream_compressed_data;
            void **stream_compressed_ptrs;
            size_t *stream_compressed_sizes;
            void **stream_uncompressed_ptrs;
            size_t *stream_uncompressed_sizes;
            nvcompStatus_t *stream_statuses;

            CHECK_CUDA(hipMalloc(&stream_compressed_data, stream_compressed_size));
            CHECK_CUDA(hipMalloc(&stream_compressed_ptrs, stream_pages * sizeof(void *)));
            CHECK_CUDA(hipMalloc(&stream_compressed_sizes, stream_pages * sizeof(size_t)));
            CHECK_CUDA(hipMalloc(&stream_uncompressed_sizes, stream_pages * sizeof(size_t)));
            CHECK_CUDA(hipMalloc(&stream_uncompressed_ptrs, stream_pages * sizeof(void *)));
            CHECK_CUDA(hipMalloc(&stream_statuses, stream_pages * sizeof(nvcompStatus_t)));

            // Setup and copy data
            void **host_compressed_ptrs = (void **)malloc(stream_pages * sizeof(void *));
            void **host_uncompressed_ptrs = (void **)malloc(stream_pages * sizeof(void *));
            size_t *host_compressed_sizes = (size_t *)malloc(stream_pages * sizeof(size_t));

            char *current_pos = stream_compressed_data;
            for (size_t i = 0; i < stream_pages; i++)
            {
                size_t page_idx = stream_start + i;
                host_compressed_ptrs[i] = current_pos;
                host_compressed_sizes[i] = compressed_data->compressed_pages[page_idx].size;
                host_uncompressed_ptrs[i] = *output_data + (page_idx * PAGE_SIZE);

                CHECK_CUDA(hipMemcpyAsync(current_pos,
                                           compressed_data->compressed_pages[page_idx].data,
                                           host_compressed_sizes[i],
                                           hipMemcpyHostToDevice,
                                           streams[stream_idx]));
                current_pos += host_compressed_sizes[i];
            }

            // Copy metadata to device
            CHECK_CUDA(hipMemcpyAsync(stream_compressed_ptrs,
                                       host_compressed_ptrs,
                                       stream_pages * sizeof(void *),
                                       hipMemcpyHostToDevice,
                                       streams[stream_idx]));

            CHECK_CUDA(hipMemcpyAsync(stream_compressed_sizes,
                                       host_compressed_sizes,
                                       stream_pages * sizeof(size_t),
                                       hipMemcpyHostToDevice,
                                       streams[stream_idx]));

            CHECK_CUDA(hipMemcpyAsync(stream_uncompressed_ptrs,
                                       host_uncompressed_ptrs,
                                       stream_pages * sizeof(void *),
                                       hipMemcpyHostToDevice,
                                       streams[stream_idx]));

            // Get decompressed sizes
            nvcompStatus_t status = nvcompBatchedLZ4GetDecompressSizeAsync(
                (const void **)stream_compressed_ptrs,
                stream_compressed_sizes,
                stream_uncompressed_sizes,
                stream_pages,
                streams[stream_idx]);

            if (status != nvcompSuccess)
                return NVCOMP_ERROR;

            // Allocate and perform decompression
            size_t temp_bytes;
            nvcompBatchedLZ4DecompressGetTempSize(stream_pages, PAGE_SIZE, &temp_bytes);
            void *stream_temp;
            CHECK_CUDA(hipMalloc(&stream_temp, temp_bytes));

            status = nvcompBatchedLZ4DecompressAsync(
                (const void **)stream_compressed_ptrs,
                stream_compressed_sizes,
                stream_uncompressed_sizes,
                stream_uncompressed_sizes,
                stream_pages,
                stream_temp,
                temp_bytes,
                stream_uncompressed_ptrs,
                stream_statuses,
                streams[stream_idx]);

            if (status != nvcompSuccess)
                return NVCOMP_ERROR;

            // Record completion event
            hipEventRecord(events[stream_idx], streams[stream_idx]);

            // Clean up stream resources
            free(host_compressed_ptrs);
            free(host_uncompressed_ptrs);
            free(host_compressed_sizes);
            hipFree(stream_compressed_data);
            hipFree(stream_compressed_ptrs);
            hipFree(stream_compressed_sizes);
            hipFree(stream_uncompressed_sizes);
            hipFree(stream_uncompressed_ptrs);
            hipFree(stream_temp);
            hipFree(stream_statuses);
        }

        // Wait for batch completion
        for (int i = 0; i < NUM_STREAMS; i++)
        {
            hipEventSynchronize(events[i]);
        }
    }

    // Final cleanup
    for (int i = 0; i < NUM_STREAMS; i++)
    {
        hipStreamDestroy(streams[i]);
        hipEventDestroy(events[i]);
    }

    return SUCCESS;
}

ErrorCode decompress_improved_naive(const CompressedData *compressed_data, char **output_data, size_t *output_size)
{
//    printf("\n=== Improved Naive Decompression ===\n");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    // printf("\n=== Starting Decom Version ===\n");
    hipEventRecord(start);

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    // Get current device for memory advice
    int current_device;
    hipGetDevice(&current_device);

    size_t num_pages = compressed_data->num_pages;
    *output_size = compressed_data->original_size;

    // Calculate total compressed size
    size_t total_compressed_size = 0;
    size_t *compressed_sizes = (size_t *)malloc(num_pages * sizeof(size_t));
    if (!compressed_sizes)
        return MEMORY_ERROR;

    for (size_t i = 0; i < num_pages; i++)
    {
        compressed_sizes[i] = compressed_data->compressed_pages[i].size;
        total_compressed_size += compressed_sizes[i];
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Initial setup: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Allocate unified memory for output early
    CHECK_CUDA(hipMallocManaged(output_data, compressed_data->original_size));

    // Add memory advice after allocation (device will access this memory)
    CHECK_CUDA(hipMemAdvise(*output_data,
                             compressed_data->original_size,
                             hipMemAdviseSetPreferredLocation,
                             current_device));

    // Allocate device buffers
    char *device_compressed_data;
    CHECK_CUDA(hipMalloc(&device_compressed_data, total_compressed_size));
    void **device_compressed_ptrs;
    CHECK_CUDA(hipMalloc(&device_compressed_ptrs, num_pages * sizeof(void *)));

    void **host_compressed_ptrs = (void **)malloc(num_pages * sizeof(void *));
    if (!host_compressed_ptrs)
    {
        hipFree(device_compressed_data);
        free(compressed_sizes);
        return MEMORY_ERROR;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Initial allocations: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Copy compressed data to GPU
    char *current_pos = device_compressed_data;
    for (size_t i = 0; i < num_pages; i++)
    {
        host_compressed_ptrs[i] = current_pos;
        CHECK_CUDA(hipMemcpyAsync(current_pos,
                                   compressed_data->compressed_pages[i].data,
                                   compressed_sizes[i],
                                   hipMemcpyHostToDevice,
                                   stream));
        current_pos += compressed_sizes[i];
    }

    CHECK_CUDA(hipMemcpyAsync(device_compressed_ptrs,
                               host_compressed_ptrs,
                               num_pages * sizeof(void *),
                               hipMemcpyHostToDevice,
                               stream));

    size_t *device_compressed_sizes;
    CHECK_CUDA(hipMalloc(&device_compressed_sizes, num_pages * sizeof(size_t)));
    CHECK_CUDA(hipMemcpyAsync(device_compressed_sizes,
                               compressed_sizes,
                               num_pages * sizeof(size_t),
                               hipMemcpyHostToDevice,
                               stream));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Data transfer to GPU: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Get decompressed sizes
    size_t *device_uncompressed_sizes;
    CHECK_CUDA(hipMalloc(&device_uncompressed_sizes, num_pages * sizeof(size_t)));

    nvcompStatus_t status = nvcompBatchedLZ4GetDecompressSizeAsync(
        (const void **)device_compressed_ptrs,
        device_compressed_sizes,
        device_uncompressed_sizes,
        num_pages,
        stream);

    if (status != nvcompSuccess)
    {
        hipFree(device_compressed_data);
        hipFree(device_compressed_ptrs);
        hipFree(device_compressed_sizes);
        hipFree(device_uncompressed_sizes);
        free(compressed_sizes);
        free(host_compressed_ptrs);
        return NVCOMP_ERROR;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Get decompressed sizes: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Setup output pointers using unified memory
    void **device_uncompressed_ptrs;
    CHECK_CUDA(hipMalloc(&device_uncompressed_ptrs, num_pages * sizeof(void *)));

    void **host_uncompressed_ptrs = (void **)malloc(num_pages * sizeof(void *));
    if (!host_uncompressed_ptrs)
    {
        hipFree(device_compressed_data);
        hipFree(device_compressed_ptrs);
        hipFree(device_compressed_sizes);
        hipFree(device_uncompressed_sizes);
        hipFree(device_uncompressed_ptrs);
        free(compressed_sizes);
        free(host_compressed_ptrs);
        return MEMORY_ERROR;
    }

    for (size_t i = 0; i < num_pages; i++)
    {
        host_uncompressed_ptrs[i] = *output_data + (i * PAGE_SIZE);
    }

    CHECK_CUDA(hipMemcpyAsync(device_uncompressed_ptrs,
                               host_uncompressed_ptrs,
                               num_pages * sizeof(void *),
                               hipMemcpyHostToDevice,
                               stream));

    size_t temp_bytes;
    nvcompBatchedLZ4DecompressGetTempSize(num_pages, PAGE_SIZE, &temp_bytes);
    void *device_temp;
    CHECK_CUDA(hipMalloc(&device_temp, temp_bytes));

    nvcompStatus_t *device_statuses;
    CHECK_CUDA(hipMalloc(&device_statuses, num_pages * sizeof(nvcompStatus_t)));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Output setup: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    status = nvcompBatchedLZ4DecompressAsync(
        (const void **)device_compressed_ptrs,
        device_compressed_sizes,
        device_uncompressed_sizes,
        device_uncompressed_sizes,
        num_pages,
        device_temp,
        temp_bytes,
        device_uncompressed_ptrs,
        device_statuses,
        stream);

    if (status != nvcompSuccess)
    {
        hipFree(device_compressed_data);
        hipFree(device_compressed_ptrs);
        hipFree(device_compressed_sizes);
        hipFree(device_uncompressed_sizes);
        hipFree(device_uncompressed_ptrs);
        hipFree(device_temp);
        hipFree(device_statuses);
        free(compressed_sizes);
        free(host_compressed_ptrs);
        free(host_uncompressed_ptrs);
        return NVCOMP_ERROR;
    }

    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Decompression: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Change preferred location to CPU before prefetching
    CHECK_CUDA(hipMemAdvise(*output_data,
                             compressed_data->original_size,
                             hipMemAdviseSetPreferredLocation,
                             hipCpuDeviceId));

    // Ensure data is available on host
    CHECK_CUDA(hipMemPrefetchAsync(*output_data,
                                    compressed_data->original_size,
                                    hipCpuDeviceId,
                                    stream));

    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Memory prefetch to host: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Cleanup
    hipFree(device_compressed_data);
    hipFree(device_compressed_ptrs);
    hipFree(device_compressed_sizes);
    hipFree(device_uncompressed_sizes);
    hipFree(device_uncompressed_ptrs);
    hipFree(device_temp);
    hipFree(device_statuses);
    free(compressed_sizes);
    free(host_compressed_ptrs);
    free(host_uncompressed_ptrs);
    hipStreamDestroy(stream);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Cleanup: %.3f ms\n", milliseconds);
    // printf("=== Profiling Complete ===\n\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return SUCCESS;
}

ErrorCode decompress_naive(const CompressedData *compressed_data, char **output_data, size_t *output_size)
{
//    printf("\n=== Naive Decompression ===\n");
    // Timing variables
    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // printf("\n=== Starting Detailed Profiling ===\n");
    hipEventRecord(start);

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    size_t num_pages = compressed_data->num_pages;
    *output_size = compressed_data->original_size;

    // Allocation timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Initial setup: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Host and device allocations
    void **host_compressed_data_per_page;
    CHECK_CUDA(hipHostMalloc(&host_compressed_data_per_page, sizeof(void *) * num_pages));
    size_t *device_compressed_numbytes_per_page;
    CHECK_CUDA(hipMalloc(&device_compressed_numbytes_per_page, sizeof(size_t) * num_pages));
    size_t *compressed_sizes = (size_t *)malloc(num_pages * sizeof(size_t));

    if (!compressed_sizes)
    {
        hipHostFree(host_compressed_data_per_page);
        hipFree(device_compressed_numbytes_per_page);
        hipStreamDestroy(stream);
        return MEMORY_ERROR;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Initial allocations: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Copy compressed data to GPU
    for (size_t i = 0; i < num_pages; i++)
    {
        compressed_sizes[i] = compressed_data->compressed_pages[i].size;
        CHECK_CUDA(hipMalloc(&host_compressed_data_per_page[i], compressed_sizes[i]));
        CHECK_CUDA(hipMemcpy(host_compressed_data_per_page[i],
                              compressed_data->compressed_pages[i].data,
                              compressed_sizes[i],
                              hipMemcpyHostToDevice));
    }

    CHECK_CUDA(hipMemcpy(device_compressed_numbytes_per_page, compressed_sizes,
                          sizeof(size_t) * num_pages, hipMemcpyHostToDevice));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Data transfer to GPU: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    free(compressed_sizes);

    // Get decompressed sizes
    size_t *device_uncompressed_numbytes_per_page;
    CHECK_CUDA(hipMalloc(&device_uncompressed_numbytes_per_page, sizeof(size_t) * num_pages));

    nvcompStatus_t status = nvcompBatchedLZ4GetDecompressSizeAsync(
        (const void **)host_compressed_data_per_page,
        device_compressed_numbytes_per_page,
        device_uncompressed_numbytes_per_page,
        num_pages,
        stream);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Get decompressed sizes: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    if (status != nvcompSuccess)
    {
        for (size_t i = 0; i < num_pages; i++)
        {
            hipFree(host_compressed_data_per_page[i]);
        }
        hipHostFree(host_compressed_data_per_page);
        hipFree(device_compressed_numbytes_per_page);
        hipFree(device_uncompressed_numbytes_per_page);
        hipStreamDestroy(stream);
        return NVCOMP_ERROR;
    }

    // Temp buffer allocation
    size_t decomp_temp_bytes;
    nvcompBatchedLZ4DecompressGetTempSize(num_pages, PAGE_SIZE, &decomp_temp_bytes);
    void *device_temp_ptr;
    CHECK_CUDA(hipMalloc(&device_temp_ptr, decomp_temp_bytes));

    nvcompStatus_t *device_statuses;
    CHECK_CUDA(hipMalloc(&device_statuses, sizeof(nvcompStatus_t) * num_pages));

    size_t *device_actual_uncompressed_numbytes_per_page;
    CHECK_CUDA(hipMalloc(&device_actual_uncompressed_numbytes_per_page, sizeof(size_t) * num_pages));

    char *device_output_data;
    CHECK_CUDA(hipMalloc(&device_output_data, compressed_data->original_size));

    void **device_uncompressed_data_per_page;
    void **host_uncompressed_data_per_page;
    CHECK_CUDA(hipMalloc(&device_uncompressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipHostMalloc(&host_uncompressed_data_per_page, sizeof(void *) * num_pages));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Temp buffer and output allocations: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Setup output pointers
    for (size_t i = 0; i < num_pages; i++)
    {
        host_uncompressed_data_per_page[i] = device_output_data + PAGE_SIZE * i;
    }
    CHECK_CUDA(hipMemcpy(device_uncompressed_data_per_page, host_uncompressed_data_per_page,
                          sizeof(void *) * num_pages, hipMemcpyHostToDevice));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Output pointer setup: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Actual decompression
    status = nvcompBatchedLZ4DecompressAsync(
        (const void **)host_compressed_data_per_page,
        device_compressed_numbytes_per_page,
        device_uncompressed_numbytes_per_page,
        device_actual_uncompressed_numbytes_per_page,
        num_pages,
        device_temp_ptr,
        decomp_temp_bytes,
        device_uncompressed_data_per_page,
        device_statuses,
        stream);

    hipStreamSynchronize(stream); // Make sure decompression is complete before timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Actual decompression: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    if (status != nvcompSuccess)
    {
        for (size_t i = 0; i < num_pages; i++)
        {
            hipFree(host_compressed_data_per_page[i]);
        }
        hipHostFree(host_compressed_data_per_page);
        hipFree(device_compressed_numbytes_per_page);
        hipFree(device_uncompressed_numbytes_per_page);
        hipFree(device_temp_ptr);
        hipFree(device_statuses);
        hipFree(device_actual_uncompressed_numbytes_per_page);
        hipFree(device_uncompressed_data_per_page);
        hipFree(device_output_data);
        hipHostFree(host_uncompressed_data_per_page);
        hipStreamDestroy(stream);
        return NVCOMP_ERROR;
    }

    // Allocate and copy result to host
    *output_data = (char *)malloc(compressed_data->original_size);
    if (!*output_data)
    {
        for (size_t i = 0; i < num_pages; i++)
        {
            hipFree(host_compressed_data_per_page[i]);
        }
        hipHostFree(host_compressed_data_per_page);
        hipFree(device_compressed_numbytes_per_page);
        hipFree(device_uncompressed_numbytes_per_page);
        hipFree(device_temp_ptr);
        hipFree(device_statuses);
        hipFree(device_actual_uncompressed_numbytes_per_page);
        hipFree(device_uncompressed_data_per_page);
        hipFree(device_output_data);
        hipHostFree(host_uncompressed_data_per_page);
        hipStreamDestroy(stream);
        return MEMORY_ERROR;
    }

    CHECK_CUDA(hipMemcpy(*output_data, device_output_data,
                          compressed_data->original_size, hipMemcpyDeviceToHost));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Copy result back to host: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Cleanup
    for (size_t i = 0; i < num_pages; i++)
    {
        hipFree(host_compressed_data_per_page[i]);
    }
    hipHostFree(host_compressed_data_per_page);
    hipFree(device_compressed_numbytes_per_page);
    hipFree(device_uncompressed_numbytes_per_page);
    hipFree(device_temp_ptr);
    hipFree(device_statuses);
    hipFree(device_actual_uncompressed_numbytes_per_page);
    hipFree(device_uncompressed_data_per_page);
    hipFree(device_output_data);
    hipHostFree(host_uncompressed_data_per_page);

    hipStreamDestroy(stream);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Cleanup: %.3f ms\n", milliseconds);
    // printf("=== Profiling Complete ===\n\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return SUCCESS;
}

// Helper function to print data
void print_data(const char *data, size_t size, const char *label)
{
    printf("\n%s (first 100 bytes):\n", label);
    for (size_t i = 0; i < (size < 100 ? size : 100); ++i)
    {
        printf("%02x ", (unsigned char)data[i]);
        if ((i + 1) % 20 == 0)
            printf("\n");
    }
    printf("\n");
}
