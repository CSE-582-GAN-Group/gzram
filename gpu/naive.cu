#include "naive.cuh"

#include <hip/hip_runtime.h>
#include "nvcomp/lz4.h"

// Error checking helper for CUDA calls
#define CHECK_CUDA(call)                                                     \
    do                                                                       \
    {                                                                        \
        hipError_t err = call;                                              \
        if (err != hipSuccess)                                              \
        {                                                                    \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err));                                \
            return CUDA_ERROR;                                               \
        }                                                                    \
    } while (0)

// Helper function to initialize empty CompressedData
CompressedData *create_compressed_data(size_t num_pages)
{
    CompressedData *data = (CompressedData *)malloc(sizeof(CompressedData));
    if (!data)
        return NULL;

    data->compressed_pages = (CompressedPage *)malloc(num_pages * sizeof(CompressedPage));
    if (!data->compressed_pages)
    {
        free(data);
        return NULL;
    }

    data->num_pages = num_pages;
    data->original_size = 0;

    // Initialize all chunks to NULL
    for (size_t i = 0; i < num_pages; i++)
    {
        data->compressed_pages[i].data = NULL;
        data->compressed_pages[i].size = 0;
    }

    return data;
}

// New helper function to create CompressedData from existing arrays
CompressedData *create_compressed_data_from_arrays(
    const char **page_data_array,   // Array of pointers to compressed page data
    const size_t *page_sizes_array, // Array of compressed page sizes
    size_t num_pages,               // Number of pages
    size_t original_size            // Original uncompressed data size
)
{
    CompressedData *data = (CompressedData *)malloc(sizeof(CompressedData));
    if (!data)
        return NULL;

    data->compressed_pages = (CompressedPage *)malloc(num_pages * sizeof(CompressedPage));
    if (!data->compressed_pages)
    {
        free(data);
        return NULL;
    }

    data->num_pages = num_pages;
    data->original_size = original_size;

    // Copy each page's data
    for (size_t i = 0; i < num_pages; i++)
    {
        data->compressed_pages[i].size = page_sizes_array[i];
        data->compressed_pages[i].data = (char *)malloc(page_sizes_array[i]);

        if (!data->compressed_pages[i].data)
        {
            // Cleanup on failure
            for (size_t j = 0; j < i; j++)
            {
                free(data->compressed_pages[j].data);
            }
            free(data->compressed_pages);
            free(data);
            return NULL;
        }
        memcpy(data->compressed_pages[i].data, page_data_array[i], page_sizes_array[i]);
    }

    return data;
}

// Create CompressedData that references existing arrays without copying
CompressedData *create_compressed_data_with_references(
    const char **page_data_array,   // Array of pointers to compressed page data
    const size_t *page_sizes_array, // Array of compressed page sizes
    size_t num_pages,               // Number of pages
    size_t original_size            // Original uncompressed data size
)
{
    CompressedData *data = (CompressedData *)malloc(sizeof(CompressedData));
    if (!data)
        return NULL;

    data->compressed_pages = (CompressedPage *)malloc(num_pages * sizeof(CompressedPage));
    if (!data->compressed_pages)
    {
        free(data);
        return NULL;
    }

    data->num_pages = num_pages;
    data->original_size = original_size;

    // Store references to each page's data instead of copying
    for (size_t i = 0; i < num_pages; i++)
    {
        data->compressed_pages[i].size = page_sizes_array[i];
        // Simply store the pointer to the original data
        data->compressed_pages[i].data = (char *)page_data_array[i];
    }

    return data;
}

// Helper function to free CompressedData
void free_compressed_data(CompressedData *data)
{
    if (!data)
        return;

    if (data->compressed_pages)
    {
        for (size_t i = 0; i < data->num_pages; i++)
        {
            free(data->compressed_pages[i].data);
        }
        free(data->compressed_pages);
    }

    free(data);
}

void cuda_initialize() {
  hipFree(0);
}

ErrorCode compress(const char *input_data, size_t in_bytes, CompressedData **output)
{
    // Create timing events
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    printf("\n=== Performance Profile ===\n");
    
    // Phase 1: Initialization
    hipEventRecord(start);
    
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    const size_t num_pages = (in_bytes + PAGE_SIZE - 1) / PAGE_SIZE;
    // Create output structure
    CompressedData *compressed_result = create_compressed_data(num_pages);
    if (!compressed_result)
    {
        hipStreamDestroy(stream);
        return MEMORY_ERROR;
    }
    compressed_result->original_size = in_bytes;

    // Allocate device input data
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("1. Initialization: %.3f ms\n", milliseconds);

    // Phase 2: Device Memory Allocation & Initial Transfer
    hipEventRecord(start);
    
    char *device_input_data;
    if (hipMalloc(&device_input_data, in_bytes) != hipSuccess)
    {
        free_compressed_data(compressed_result);
        hipStreamDestroy(stream);
        return CUDA_ERROR;
    }

    // Copy input data to device
    CHECK_CUDA(hipMemcpyAsync(device_input_data, input_data, in_bytes,
                               hipMemcpyHostToDevice, stream));
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("2. Initial Device Allocation & Transfer: %.3f ms\n", milliseconds);

    // Phase 3: Page Management Setup
    hipEventRecord(start);
    
    // Find bytes per page on input data
    size_t *host_uncompressed_numbytes_per_page;
    CHECK_CUDA(hipHostMalloc(&host_uncompressed_numbytes_per_page, sizeof(size_t) * num_pages));
    for (size_t i = 0; i < num_pages; ++i)
    {
        if (i + 1 < num_pages)
        {
            host_uncompressed_numbytes_per_page[i] = PAGE_SIZE;
        }
        else
        {
            host_uncompressed_numbytes_per_page[i] = in_bytes - (PAGE_SIZE * i);
        }
    }

    // Fill in the pointers to the input data for each page
    void **host_uncompressed_data_per_page;
    CHECK_CUDA(hipHostMalloc(&host_uncompressed_data_per_page, sizeof(void *) * num_pages));
    for (size_t i = 0; i < num_pages; ++i)
    {
        host_uncompressed_data_per_page[i] = device_input_data + PAGE_SIZE * i;
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("3. Page Management Setup: %.3f ms\n", milliseconds);

    // Phase 4: Device Memory Setup for Pages
    hipEventRecord(start);
    
    // Copy pointers of sizes and data to device
    size_t *device_uncompressed_numbytes_per_page;
    void **device_uncompressed_data_per_page;
    CHECK_CUDA(hipMalloc(&device_uncompressed_numbytes_per_page, sizeof(size_t) * num_pages));
    CHECK_CUDA(hipMalloc(&device_uncompressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMemcpyAsync(device_uncompressed_numbytes_per_page, host_uncompressed_numbytes_per_page,
                               sizeof(size_t) * num_pages, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(device_uncompressed_data_per_page, host_uncompressed_data_per_page,
                               sizeof(void *) * num_pages, hipMemcpyHostToDevice, stream));
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("4. Device Page Setup: %.3f ms\n", milliseconds);


    // Phase 5: Compression Buffer Setup - OPTIMIZED
    hipEventRecord(start);
    
    size_t temp_bytes;
    nvcompBatchedLZ4CompressGetTempSize(num_pages, PAGE_SIZE,
                                       nvcompBatchedLZ4DefaultOpts, &temp_bytes);
    void *device_temp_ptr;
    CHECK_CUDA(hipMalloc(&device_temp_ptr, temp_bytes));

    // Allocate a single large buffer instead of many small ones
    void **host_compressed_data_per_page;
    void **device_compressed_data_per_page;
    size_t *device_compressed_numbytes_per_page;
    CHECK_CUDA(hipHostMalloc(&host_compressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMalloc(&device_compressed_data_per_page, sizeof(void *) * num_pages));
    CHECK_CUDA(hipMalloc(&device_compressed_numbytes_per_page, sizeof(size_t) * num_pages));

    size_t max_out_bytes;
    nvcompBatchedLZ4CompressGetMaxOutputChunkSize(PAGE_SIZE,
                                                 nvcompBatchedLZ4DefaultOpts,
                                                 &max_out_bytes);
    
    // Allocate one large buffer for all pages
    void* compressed_data_buffer;
    CHECK_CUDA(hipMalloc(&compressed_data_buffer, max_out_bytes * num_pages));
    
    // Set up pointers into the buffer
    for (size_t i = 0; i < num_pages; ++i) {
        host_compressed_data_per_page[i] = (char*)compressed_data_buffer + (i * max_out_bytes);
    }
    CHECK_CUDA(hipMemcpyAsync(device_compressed_data_per_page, host_compressed_data_per_page,
                              sizeof(void *) * num_pages, hipMemcpyHostToDevice, stream));
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("5. Compression Buffer Setup: %.3f ms\n", milliseconds);

    // Phase 6: Main Compression
    hipEventRecord(start);

    // Compress the data
    nvcompStatus_t comp_res = nvcompBatchedLZ4CompressAsync(
        device_uncompressed_data_per_page,
        device_uncompressed_numbytes_per_page,
        PAGE_SIZE,
        num_pages,
        device_temp_ptr,
        temp_bytes,
        device_compressed_data_per_page,
        device_compressed_numbytes_per_page,
        nvcompBatchedLZ4DefaultOpts,
        stream);

    if (comp_res != nvcompSuccess)
    {
        hipFree(device_input_data);
        hipHostFree(host_uncompressed_numbytes_per_page);
        hipHostFree(host_uncompressed_data_per_page);
        hipFree(device_uncompressed_numbytes_per_page);
        hipFree(device_uncompressed_data_per_page);
        hipFree(device_temp_ptr);
        for (size_t i = 0; i < num_pages; i++)
        {
            hipFree(host_compressed_data_per_page[i]);
        }
        hipHostFree(host_compressed_data_per_page);
        hipFree(device_compressed_data_per_page);
        hipFree(device_compressed_numbytes_per_page);
        free_compressed_data(compressed_result);
        hipStreamDestroy(stream);
        return NVCOMP_ERROR;
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("6. Main Compression: %.3f ms\n", milliseconds);

    // Phase 7: Result Collection - OPTIMIZED
    hipEventRecord(start);
    
    // Allocate a single host buffer for all compressed data
    char* host_compressed_buffer;
    CHECK_CUDA(hipHostMalloc(&host_compressed_buffer, max_out_bytes * num_pages));
    
    // Get all compressed sizes in one transfer
    size_t *compressed_sizes = (size_t *)malloc(num_pages * sizeof(size_t));
    if (!compressed_sizes) {
        return MEMORY_ERROR;
    }

    CHECK_CUDA(hipMemcpyAsync(compressed_sizes, device_compressed_numbytes_per_page,
                              sizeof(size_t) * num_pages, hipMemcpyDeviceToHost, stream));
    
    // Copy all compressed data in one large transfer
    CHECK_CUDA(hipMemcpyAsync(host_compressed_buffer, compressed_data_buffer,
                              max_out_bytes * num_pages, hipMemcpyDeviceToHost, stream));
    
    // Wait for transfers to complete
    CHECK_CUDA(hipStreamSynchronize(stream));
    
    // Set up the output structure (now using CPU memory)
    for (size_t i = 0; i < num_pages; i++) {
        compressed_result->compressed_pages[i].size = compressed_sizes[i];
        compressed_result->compressed_pages[i].data = (char *)malloc(compressed_sizes[i]);
        if (!compressed_result->compressed_pages[i].data) {
            free(compressed_sizes);
            hipHostFree(host_compressed_buffer);
            return MEMORY_ERROR;
        }
        
        // Copy from pinned buffer to final destination (CPU memory copy, no CUDA involved) TODO: discuss with nolan, we can copy directly to the right region
        memcpy(compressed_result->compressed_pages[i].data,
               host_compressed_buffer + (i * max_out_bytes),
               compressed_sizes[i]);
    }

    free(compressed_sizes);
    hipHostFree(host_compressed_buffer);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("7. Result Collection: %.3f ms\n", milliseconds);

    // Phase 8: Cleanup - OPTIMIZED
    hipEventRecord(start);
    
    // Single large free instead of many small ones
    hipFree(compressed_data_buffer);
    hipFree(device_input_data);
    hipHostFree(host_uncompressed_numbytes_per_page);
    hipHostFree(host_uncompressed_data_per_page);
    hipFree(device_uncompressed_numbytes_per_page);
    hipFree(device_uncompressed_data_per_page);
    hipFree(device_temp_ptr);
    hipHostFree(host_compressed_data_per_page);
    hipFree(device_compressed_data_per_page);
    hipFree(device_compressed_numbytes_per_page);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("8. Cleanup: %.3f ms\n", milliseconds);

    // Cleanup timing events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipStreamDestroy(stream);

    *output = compressed_result;
    return SUCCESS;
}

ErrorCode decompress(const CompressedData *compressed_data, char **output_data, size_t *output_size) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    printf("\n=== Starting Unified Memory Version ===\n");
    hipEventRecord(start);

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    // Get current device for memory advice
    int current_device;
    hipGetDevice(&current_device);

    size_t num_pages = compressed_data->num_pages;
    *output_size = compressed_data->original_size;

    // Calculate total compressed size
    size_t total_compressed_size = 0;
    size_t *compressed_sizes = (size_t *)malloc(num_pages * sizeof(size_t));
    if (!compressed_sizes) return MEMORY_ERROR;

    for (size_t i = 0; i < num_pages; i++) {
        compressed_sizes[i] = compressed_data->compressed_pages[i].size;
        total_compressed_size += compressed_sizes[i];
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Initial setup: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Allocate unified memory for output early
    CHECK_CUDA(hipMallocManaged(output_data, compressed_data->original_size));

    // Add memory advice after allocation
    CHECK_CUDA(hipMemAdvise(*output_data, 
                            compressed_data->original_size,
                            hipMemAdviseSetPreferredLocation, 
                            current_device));

    // Allocate device buffers
    char *device_compressed_data;
    CHECK_CUDA(hipMalloc(&device_compressed_data, total_compressed_size));
    void **device_compressed_ptrs;
    CHECK_CUDA(hipMalloc(&device_compressed_ptrs, num_pages * sizeof(void*)));

    void **host_compressed_ptrs = (void**)malloc(num_pages * sizeof(void*));
    if (!host_compressed_ptrs) {
        hipFree(device_compressed_data);
        free(compressed_sizes);
        return MEMORY_ERROR;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Initial allocations: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Copy compressed data to GPU
    char* current_pos = device_compressed_data;
    for (size_t i = 0; i < num_pages; i++) {
        host_compressed_ptrs[i] = current_pos;
        CHECK_CUDA(hipMemcpyAsync(current_pos,
                                compressed_data->compressed_pages[i].data,
                                compressed_sizes[i],
                                hipMemcpyHostToDevice,
                                stream));
        current_pos += compressed_sizes[i];
    }

    CHECK_CUDA(hipMemcpyAsync(device_compressed_ptrs, 
                            host_compressed_ptrs,
                            num_pages * sizeof(void*),
                            hipMemcpyHostToDevice,
                            stream));

    size_t *device_compressed_sizes;
    CHECK_CUDA(hipMalloc(&device_compressed_sizes, num_pages * sizeof(size_t)));
    CHECK_CUDA(hipMemcpyAsync(device_compressed_sizes,
                            compressed_sizes,
                            num_pages * sizeof(size_t),
                            hipMemcpyHostToDevice,
                            stream));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Data transfer to GPU: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Get decompressed sizes
    size_t *device_uncompressed_sizes;
    CHECK_CUDA(hipMalloc(&device_uncompressed_sizes, num_pages * sizeof(size_t)));

    nvcompStatus_t status = nvcompBatchedLZ4GetDecompressSizeAsync(
        (const void **)device_compressed_ptrs,
        device_compressed_sizes,
        device_uncompressed_sizes,
        num_pages,
        stream);

    if (status != nvcompSuccess) {
        hipFree(device_compressed_data);
        hipFree(device_compressed_ptrs);
        hipFree(device_compressed_sizes);
        hipFree(device_uncompressed_sizes);
        free(compressed_sizes);
        free(host_compressed_ptrs);
        return NVCOMP_ERROR;
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Get decompressed sizes: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Setup output pointers using unified memory
    void **device_uncompressed_ptrs;
    CHECK_CUDA(hipMalloc(&device_uncompressed_ptrs, num_pages * sizeof(void*)));

    void **host_uncompressed_ptrs = (void**)malloc(num_pages * sizeof(void*));
    if (!host_uncompressed_ptrs) {
        hipFree(device_compressed_data);
        hipFree(device_compressed_ptrs);
        hipFree(device_compressed_sizes);
        hipFree(device_uncompressed_sizes);
        hipFree(device_uncompressed_ptrs);
        free(compressed_sizes);
        free(host_compressed_ptrs);
        return MEMORY_ERROR;
    }

    for (size_t i = 0; i < num_pages; i++) {
        host_uncompressed_ptrs[i] = *output_data + (i * PAGE_SIZE);
    }

    CHECK_CUDA(hipMemcpyAsync(device_uncompressed_ptrs,
                            host_uncompressed_ptrs,
                            num_pages * sizeof(void*),
                            hipMemcpyHostToDevice,
                            stream));

    size_t temp_bytes;
    nvcompBatchedLZ4DecompressGetTempSize(num_pages, PAGE_SIZE, &temp_bytes);
    void *device_temp;
    CHECK_CUDA(hipMalloc(&device_temp, temp_bytes));

    nvcompStatus_t *device_statuses;
    CHECK_CUDA(hipMalloc(&device_statuses, num_pages * sizeof(nvcompStatus_t)));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Output setup: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    status = nvcompBatchedLZ4DecompressAsync(
        (const void **)device_compressed_ptrs,
        device_compressed_sizes,
        device_uncompressed_sizes,
        device_uncompressed_sizes,
        num_pages,
        device_temp,
        temp_bytes,
        device_uncompressed_ptrs,
        device_statuses,
        stream);

    if (status != nvcompSuccess) {
        hipFree(device_compressed_data);
        hipFree(device_compressed_ptrs);
        hipFree(device_compressed_sizes);
        hipFree(device_uncompressed_sizes);
        hipFree(device_uncompressed_ptrs);
        hipFree(device_temp);
        hipFree(device_statuses);
        free(compressed_sizes);
        free(host_compressed_ptrs);
        free(host_uncompressed_ptrs);
        return NVCOMP_ERROR;
    }

    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Decompression: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Change preferred location to CPU before prefetching
    CHECK_CUDA(hipMemAdvise(*output_data,
                            compressed_data->original_size,
                            hipMemAdviseSetPreferredLocation,
                            hipCpuDeviceId));

    // Ensure data is available on host
    CHECK_CUDA(hipMemPrefetchAsync(*output_data, 
                                compressed_data->original_size,
                                hipCpuDeviceId,
                                stream));

    hipStreamSynchronize(stream);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Memory prefetch to host: %.3f ms\n", milliseconds);
    hipEventRecord(start);

    // Cleanup
    hipFree(device_compressed_data);
    hipFree(device_compressed_ptrs);
    hipFree(device_compressed_sizes);
    hipFree(device_uncompressed_sizes);
    hipFree(device_uncompressed_ptrs);
    hipFree(device_temp);
    hipFree(device_statuses);
    free(compressed_sizes);
    free(host_compressed_ptrs);
    free(host_uncompressed_ptrs);
    hipStreamDestroy(stream);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Cleanup: %.3f ms\n", milliseconds);
    printf("=== Profiling Complete ===\n\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return SUCCESS;
}

// Helper function to print data
void print_data(const char *data, size_t size, const char *label)
{
    printf("\n%s (first 100 bytes):\n", label);
    for (size_t i = 0; i < (size < 100 ? size : 100); ++i)
    {
        printf("%02x ", (unsigned char)data[i]);
        if ((i + 1) % 20 == 0)
            printf("\n");
    }
    printf("\n");
}

